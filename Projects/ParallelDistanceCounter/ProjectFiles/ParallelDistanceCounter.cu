
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "omp.h"
#include <iostream>

//See values of N in assignment instructions.
#define N 100000
//Do not change the seed, or your answer will not be correct
#define SEED 72

//For GPU implementation
#define BLOCKSIZE 1024

using namespace std;

struct pointData{
double x;
double y;
};


void generateDataset(struct pointData * data);
__global__ void distanceCalc( struct pointData * data, unsigned int * count, double * epsilon );


int main(int argc, char *argv[])
{
	//Read epsilon distance from command line
	if (argc!=2)
	{
		printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
		return 0;
	}
	
	
	char inputEpsilon[20];
	strcpy(inputEpsilon,argv[1]);
	double epsilon=atof(inputEpsilon);
	
	

	//generate dataset:
	struct pointData * data;
	data=(struct pointData*)malloc(sizeof(struct pointData)*N);
	printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(1024.0*1024.0));
	generateDataset(data);


	omp_set_num_threads(1);


	double tstart=omp_get_wtime();

	hipError_t errCode = hipSuccess;

	if( errCode != hipSuccess )
	{
		cout << "\nLast error: " << errCode << endl;
	}
	
	struct pointData * dev_data;
	unsigned int * count;
	unsigned int * dev_count;
	double * dev_epsilon;

	count = (unsigned int *)malloc(sizeof(unsigned int));
	dev_count = (unsigned int *)malloc(sizeof(unsigned int));
	*count = N;

	dev_epsilon = (double *)malloc(sizeof(double));

	// allocate data on device
	errCode = hipMalloc((struct pointData **)&dev_data, sizeof(struct pointData)*N);
	if( errCode != hipSuccess )
	{
		cout << "\nError: dev_data allocation error with code " << errCode << endl;
	}

	errCode = hipMalloc((unsigned int **)&dev_count, sizeof(unsigned int));
	if( errCode != hipSuccess )
	{
		cout << "\nError: dev_count allocation error with code " << errCode << endl;
	}

	errCode = hipMalloc((double **)&dev_epsilon, sizeof(double));
        if( errCode != hipSuccess )
        {
                cout << "\nError: dev_epsilon allocation error with code " << errCode << endl;
        }

	// copy data over to the device
	errCode = hipMemcpy( dev_data, data, sizeof(struct pointData)*N, hipMemcpyHostToDevice );
	if( errCode != hipSuccess )
	{
		cout << "\nError: dev_data copy in error with code " << errCode << endl;
	}

	errCode = hipMemcpy( dev_count, count, sizeof(unsigned int), hipMemcpyHostToDevice );
	if( errCode != hipSuccess )
	{
		cout << "\nError: dev_count copy in error with code " << errCode << endl;
	}

	errCode = hipMemcpy( dev_epsilon, &epsilon, sizeof(double), hipMemcpyHostToDevice );
        if( errCode != hipSuccess )
        {
                cout << "\nError: dev_epsilon copy in error with code " << errCode << endl;
        }


	// calculate blocks
	const unsigned int totalBlocks = ceil(N*1.0/BLOCKSIZE);


	// execute kernel
	double tkstart = omp_get_wtime();
	distanceCalc<<<totalBlocks, BLOCKSIZE>>>(dev_data, dev_count, dev_epsilon);

	hipDeviceSynchronize();
	if( errCode != hipSuccess )
	{
		cout << "\nErrorafter kernel launch " << errCode << endl;
	}

	double tkend = omp_get_wtime();

	// transfer count back to host
	errCode = hipMemcpy( count, dev_count, sizeof(unsigned int), hipMemcpyDeviceToHost );
	if( errCode != hipSuccess )
	{
		cout << "\nError: dev_count copy out error with code " << errCode << endl;
	}

	printf("\nTotal count: %d", *count);

	double tend=omp_get_wtime();
	
	printf("\nTotal time (s): %f",tend-tstart);
	printf("\nKernel time (s): %f\n",tkend-tkstart);


	free(data);
	printf("\n");
	return 0;
}


//Generates the dataset using the given seed defined at the top
void generateDataset(struct pointData * data)
{

	//seed RNG
	srand(SEED);


	for (unsigned int i=0; i<N; i++){
		data[i].x=1000.0*((double)(rand()) / RAND_MAX);	
		data[i].y=1000.0*((double)(rand()) / RAND_MAX);	
	}
	

}

//Gets called by each thread to handle the data being passed to each specific thread
__global__ void distanceCalc( struct pointData * data, unsigned int * count, double * epsilon )
{
	unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

	if( tid >= N )
	{
		return;
	}

	unsigned int index;
	double xDiff, yDiff, distance;

	for( index = tid+1; index < N; index++ )
	{
		xDiff = data[tid].x - data[index].x;
		yDiff = data[tid].y - data[index].y;
		distance = sqrt( (xDiff * xDiff) + (yDiff * yDiff) );
		if( distance <= (*epsilon) )
		{
			atomicAdd( count, int(2) );
		}
	}
	return;
}
